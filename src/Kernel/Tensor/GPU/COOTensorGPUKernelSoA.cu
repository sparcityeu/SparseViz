#include "COOTensorGPUKernelSoA.h"
#include "GPUKernels.cuh"
#include "hip/hip_runtime.h"
#include "SparseTensorCOO.h"
#include "helpers.h"

bool COOTensorGPUKernelSoA::init(const SparseTensor &A)
{
    const SparseTensorCOO& tensorCOO = *getCOOFormat(&A);

    h_output = 0.0;
    h_nonzeros = tensorCOO.getStorage();
    dims = A.getDims();
    nnzcount = tensorCOO.getNNZ();
    order = A.getOrder();

    vType* dim_offsets = new vType[order + 1];
    dim_offsets[0] = 0;
    for(int i = 0; i < order; i++) dim_offsets[i+1] = dims[i];

    h_arrays = new float[dim_offsets[order]];
    for(vType i = 0; i < dim_offsets[order]; i++) {
        h_arrays[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
    }

    hipMalloc(&d_arrays, sizeof(float) * dim_offsets[order]);
    hipMemcpy(d_arrays, h_arrays,  sizeof(float) * dim_offsets[order], hipMemcpyHostToDevice);

    vType* nonzeros_soa = new vType[nnzcount * order];
    for(eType i = 0; i < nnzcount; i++) {
        for(int j = 0; j < order; j++) {
            nonzeros_soa[(j * nnzcount) + i] = h_nonzeros[(i * order) + j] + dim_offsets[j];    
        }
    }

    gpuErrchk( hipMalloc(&d_nonzeros, nnzcount * order * sizeof(vType)));
    gpuErrchk( hipMemcpy(d_nonzeros, nonzeros_soa, nnzcount * order * sizeof(vType), hipMemcpyHostToDevice));
    
    gpuErrchk( hipMalloc(&d_output, sizeof(float)));
    gpuErrchk( hipMemcpy(d_output, &h_output, sizeof(float), hipMemcpyHostToDevice));
    
    delete [] nonzeros_soa;
    delete [] dim_offsets;
    return true;
}

COOTensorGPUKernelSoA::~COOTensorGPUKernelSoA()
{
    gpuErrchk( hipFree(d_arrays));
    delete [] h_arrays;
    
    gpuErrchk( hipFree(d_nonzeros));
    gpuErrchk( hipFree(d_output));

}

void COOTensorGPUKernelSoA::hostFunction(const SparseTensor &A, int iterNumber, int gridSize, int blockSize)
{
    for(int i = 0; i < 10; i++) {
        cooTensorAoS<<<gridSize, blockSize>>>(d_nonzeros, nnzcount, order, d_arrays, d_output);
    }
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}

#include "COOTensorGPUKernelAoS.h"
#include "GPUKernels.cuh"
#include "hip/hip_runtime.h"
#include "SparseTensorCOO.h"
#include "helpers.h"

bool COOTensorGPUKernelAoS::init(const SparseTensor &A)
{
    const SparseTensorCOO& tensorCOO = *getCOOFormat(&A);

    h_output = 0.0f;
    h_nonzeros = tensorCOO.getStorage();
    dims = A.getDims();
    nnzcount = tensorCOO.getNNZ();
    order = A.getOrder();

    vType* dim_offsets = new vType[order + 1];
    dim_offsets[0] = 0;
    for(int i = 0; i < order; i++) dim_offsets[i+1] = dims[i];

    h_arrays = new float[dim_offsets[order]];
    for(vType i = 0; i < dim_offsets[order]; i++) {
        h_arrays[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
    }

    hipMalloc(&d_arrays, sizeof(float) * dim_offsets[order]);
    hipMemcpy(d_arrays, h_arrays,  sizeof(float) * dim_offsets[order], hipMemcpyHostToDevice);

    vType* nonzeros_soa = new vType[nnzcount * order];
    for(eType i = 0; i < nnzcount; i++) {
        for(int j = 0; j < order; j++) {
            nonzeros_soa[(i * order) + j] = h_nonzeros[(i * order) + j] + dim_offsets[j];    
        }
    }

    hipMalloc(&d_nonzeros, nnzcount * order * sizeof(vType));
    hipMemcpy(d_nonzeros, nonzeros_soa, nnzcount * order * sizeof(vType), hipMemcpyHostToDevice);
    
    hipMalloc(&d_output, sizeof(float));
    hipMemcpy(d_output, &h_output, sizeof(float), hipMemcpyHostToDevice);

    delete [] dim_offsets;
    delete [] nonzeros_soa;
    
    return true;
}

COOTensorGPUKernelAoS::~COOTensorGPUKernelAoS()
{
    hipFree(d_arrays);
    delete [] h_arrays;
    
    hipFree(d_nonzeros);
    hipFree(d_output);

}

void COOTensorGPUKernelAoS::hostFunction(const SparseTensor &A, int iterNumber, int gridSize, int blockSize)
{
    for(int i = 0; i < 10; i++) {
        cooTensorAoS<<<gridSize, blockSize>>>(d_nonzeros, nnzcount, order, d_arrays, d_output);
    }
    hipDeviceSynchronize();
}

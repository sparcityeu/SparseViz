#include "hip/hip_runtime.h"
#include "COOTensorGPUKernelAoS.h"
#include "SparseTensorCOO.h"
#include "helpers.h"


__global__ void cooTensorAoS(const vType* storage, eType nnzcount, int order, const float* input_vals, float* output) {
    int no_threads = blockDim.x * gridDim.x;
    eType tid = threadIdx.x + blockDim.x * blockIdx.x;
    float sum = 0;
    for(eType i = tid; i < nnzcount; i += no_threads) {
        float temp = 1;
        for(int m = 0; m < order; m++) {
            temp *= input_vals[storage[i * order + m]];
        }
        sum += temp;
    }
    atomicAdd(output, sum);
}


bool COOTensorGPUKernelAoS::init(const SparseTensor &A)
{
    const SparseTensorCOO& tensorCOO = dynamic_cast<const SparseTensorCOO&>(A);

    h_output = 0.0f;
    h_nonzeros = tensorCOO.getStorage();
    dims = A.getDims();
    nnzcount = tensorCOO.getNNZ();
    order = A.getOrder();

    vType* dim_offsets = new vType[order + 1];
    dim_offsets[0] = 0;
    for(int i = 0; i < order; i++) dim_offsets[i+1] = dims[i];

    h_arrays = new float[dim_offsets[order]];
    for(vType i = 0; i < dim_offsets[order]; i++) {
        h_arrays[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
    }

    hipMalloc(&d_arrays, sizeof(float) * dim_offsets[order]);
    hipMemcpy(d_arrays, h_arrays,  sizeof(float) * dim_offsets[order], hipMemcpyHostToDevice);

    vType* nonzeros_soa = new vType[nnzcount * order];
    for(eType i = 0; i < nnzcount; i++) {
        for(int j = 0; j < order; j++) {
            nonzeros_soa[(i * order) + j] = h_nonzeros[(i * order) + j] + dim_offsets[j];    
        }
    }

    hipMalloc(&d_nonzeros, nnzcount * order * sizeof(vType));
    hipMemcpy(d_nonzeros, nonzeros_soa, nnzcount * order * sizeof(vType), hipMemcpyHostToDevice);
    
    hipMalloc(&d_output, sizeof(float));
    hipMemcpy(d_output, &h_output, sizeof(float), hipMemcpyHostToDevice);

    delete [] dim_offsets;
    delete [] nonzeros_soa;
    
    return true;
}

COOTensorGPUKernelAoS::~COOTensorGPUKernelAoS()
{
    hipFree(d_arrays);
    delete [] h_arrays;
    
    hipFree(d_nonzeros);
    hipFree(d_output);

}

void COOTensorGPUKernelAoS::hostFunction(const SparseTensor &A, int iterNumber, int gridSize, int blockSize, int sharedMemorySize)
{
    for(int i = 0; i < 10; i++) {
        cooTensorAoS<<<gridSize, blockSize, sharedMemorySize>>>(d_nonzeros, nnzcount, order, d_arrays, d_output);
    }
    hipDeviceSynchronize();
}
